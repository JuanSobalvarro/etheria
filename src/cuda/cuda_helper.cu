#include "cuda/cuda_helper.cuh"
#include <cstdio>
#include <stdexcept>

namespace eth::cuda 
{

bool isCUDAAvailable() 
{
    int deviceCount = 0;
    hipError_t error_id = hipGetDeviceCount(&deviceCount);
    return (error_id == hipSuccess && deviceCount > 0);
}

int numberCUDADevices() 
{
    int deviceCount = 0;
    hipError_t error_id = hipGetDeviceCount(&deviceCount);
    return (error_id == hipSuccess) ? deviceCount : 0;
}

bool isCUDACompatible(int device_id) 
{
    int deviceCount = 0;
    if (hipGetDeviceCount(&deviceCount) != hipSuccess || device_id < 0 || device_id >= deviceCount)
        return false;

    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, device_id);
    return deviceProp.major >= 3;
}

std::vector<std::string> listCUDADevices() 
{
    std::vector<std::string> deviceNames;
    int deviceCount = 0;
    if (hipGetDeviceCount(&deviceCount) != hipSuccess) return deviceNames;

    for (int i = 0; i < deviceCount; ++i) {
        hipDeviceProp_t deviceProp;
        hipGetDeviceProperties(&deviceProp, i);
        deviceNames.push_back(deviceProp.name);
    }
    return deviceNames;
}

void checkCuda(hipError_t err, const char* msg) 
{
    if (err != hipSuccess) {
        if (msg) fprintf(stderr, "CUDA error (%s): %s\n", msg, hipGetErrorString(err));
        else fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(err));
        std::abort();
    }
}

void setDevice(int device_id) 
{
    int device_count = 0;
    hipGetDeviceCount(&device_count);
    if (device_id < 0 || device_id >= device_count)
        throw std::runtime_error("Invalid CUDA device ID");
    hipSetDevice(device_id);
}

} // namespace eth::cuda
